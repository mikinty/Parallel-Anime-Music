#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "training.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

//Transforms 2 (noteTone, noteLength) pairs to a matrix index
static inline int findNoteCell(int curTone, int curDur, int prevTone1, int prevDur1, int prevTone2, int prevDur2)
{
  // current note 
  int col = curTone * curDur - 1;

  //If previous tones are chords, get top note and find closest
  if (prevTone1 > NUM_TONES){
      prevTone1 = prevTone1 / 144; //get top chord note
      prevTone1 = curTone - (curTone % 12) + prevTone1;
  }
  if (prevTone2 > NUM_TONES){
      prevTone2 = prevTone2 / 144; //get top chord note
      prevTone2 = curTone - (curTone % 12) + prevTone2;
  }

  int row = prevTone1 * prevDur1 * prevTone2 * prevDur2 - 1;

  return row * NUM_NOTES + col;
}

//Gets flattened matrix index
static inline int findChordCell(int curTone, int prevTone){
	
  if (prevTone > NUM_TONES){
        prevTone = prevTone - NUM_TONES - 1; //shift chord down
  }
  
  return prevTone * NUM_CHORDS + (curTone - NUM_TONES - 1);
}

__global__ void CountSection(sound_t* deviceS,int sLength,sound_t* deviceB,int bLength,float* deviceHigh,float* deviceLow,float* deviceChord)
{
    sound_t* part;
    int start;
    int end;
    __shared__ float melodyM[NUM_NOTES * NUM_NOTES * NUM_NOTES];

    //initialize shared mem
    start = threadIdx.x * (NUM_NOTES * NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
    end = start + (NUM_NOTES * NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
    for (int mIndex = start; mIndex < end; mIndex++){
        if (mIndex > NUM_NOTES * NUM_NOTES * NUM_NOTES)
            break;
        melodyM[mIndex] = 0;
    }

    if (blockIdx.x == 0){
        part = deviceS;
        start = threadIdx.x * (sLength / NUM_THREADS) + 2;
        if (threadIdx.x == NUM_THREADS - 1){
            end = sLength;
        }
        else{
            end = start + sLength/NUM_THREADS;
        }
    }
    else{
        part = deviceB;
        start = threadIdx.x * (bLength / NUM_THREADS) + 2;
        if (threadIdx.x == NUM_THREADS - 1){
            end = bLength;
        }
        else{
            end = start + bLength/NUM_THREADS;
        }
    }

    int prevTone1 = part[start-2].tone;
    int prevDur1 = part[start-2].duration;
    int curTone = part[start-1].tone;
    int curDur = part[start-1].duration;

    __syncthreads();

    for (int noteIndex = start; noteIndex < end; noteIndex++){    	
    	int prevTone2 = prevTone1;
    	int prevDur2 = prevDur1;
    	prevTone1 = curTone;
        prevDur1 = curDur;
        curTone = part[noteIndex].tone;
        curDur = part[noteIndex].duration;

        int cell;
        if (curTone > NUM_TONES) { //insert into chord matrix
            cell = findChordCell(curTone, prevTone1);
            atomic_add(deviceChord[cell], 1);
        }
        else { 
          //insert into melody note matrix
          cell = findNoteCell(curTone, curDur, prevTone1, prevDur1, prevTone2, prevDur2);
          atomic_add(melodyM[cell], 1);
        }
    }

    __syncthreads();

  // copy shared mem matrix into the global
    start = threadIdx.x * (NUM_NOTES * NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
    end = start + (NUM_NOTES * NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
    for (int mIndex = start; mIndex < end; mIndex++){
        if (mIndex > NUM_NOTES * NUM_NOTES * NUM_NOTES)
            break;
        if (blockIdx.x == 0)
            deviceHigh[mIndex] += melodyM[mIndex];
        else
            deviceLow[mIndex] += melodyM[mIndex];
    }
    
}

void initCuda(float* deviceHigh, float* deviceLow, float* deviceChord, sound_t* deviceS, sound_t* deviceB)
{
  hipMalloc((void**)&deviceHigh, sizeof(float) * NUM_NOTES * NUM_NOTES * NUM_NOTES);
  hipMalloc((void**)&deviceLow, sizeof(float) * NUM_NOTES * NUM_NOTES * NUM_NOTES);
  hipMalloc((void**)&deviceChord, sizeof(float) * NUM_CHORDS * NUM_CHORDS); 

  hipMemset(deviceChord, 0, sizeof(float) * NUM_CHORDS * NUM_CHORDS);
  hipMemset(deviceLow, 0, sizeof(float) * NUM_NOTES * NUM_NOTES * NUM_NOTES);
  hipMemset(deviceHigh, 0, sizeof(float) * NUM_NOTES * NUM_NOTES * NUM_NOTES);

  __syncthreads();
}

// TODO: FREE FUNCTION / CLEANUP
void countTransitionsCuda(sound_t* soprano, int sLength, sound_t* bass, int bLength, float* deviceHigh, float* deviceLow, float* deviceChord){

  sound_t* deviceS;
  sound_t* deviceB;
  hipMalloc((void **)&deviceS, sizeof(sound_t) * sLength);
  hipMemcpy(deviceS, soprano, sizeof(sound_t) * sLength, hipMemcpyHostToDevice);
  hipMalloc((void **)&deviceB, sizeof(sound_t) * bLength);
  hipMemcpy(deviceB, bass, sizeof(sound_t) * bLength, hipMemcpyHostToDevice);

  CountSection<<<NUM_THREADS, 2>>>(deviceS, sLength, deviceB, bLength, deviceHigh, deviceLow, deviceChord);

  hipDeviceSynchronize();

  // free file buffers
  hipFree(deviceS);
  hipFree(deviceB);
}

__global__ void normalizeRow(float* deviceHigh,float* deviceLow,float* deviceChord){
    int start;
    int end;
    if(blockIdx.x == 0){
        start = threadIdx.x * (NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
        end = start + (NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
        if (end > NUM_NOTES * NUM_NOTES){
            end = NUM_NOTES * NUM_NOTES;
        }
        for (int i = start; i < end; i++){
            int sum = 0;
            for(int j = 0; j < NUM_NOTES; j++){
                sum = sum + deviceHigh[i * NUM_NOTES + j];
            }
            for (int j = 0; j < NUM_NOTES; j++){
                deviceHigh[i * NUM_NOTES + j] = deviceHigh[i * NUM_NOTES + j] / sum;
            }
        }
    }
    else if (blockIdx.x == 1){
        start = threadIdx.x * (NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
        end = start + (NUM_NOTES * NUM_NOTES + NUM_THREADS - 1)/NUM_THREADS;
        if (end > NUM_NOTES * NUM_NOTES){
            end = NUM_NOTES * NUM_NOTES;
        }
        for (int i = start; i < end; i++){
            int sum = 0;
            for(int j = 0; j < NUM_NOTES; j++){
                sum = sum + deviceLow[i * NUM_NOTES + j];
            }
            for (int j = 0; j < NUM_NOTES; j++){
                deviceLow[i * NUM_NOTES + j] = deviceLow[i * NUM_NOTES + j] / sum;
            }
        }
    }
    else{
        start = threadIdx.x * (NUM_CHORDS + NUM_THREADS - 1)/NUM_THREADS;
        end = start + (NUM_NOTES * NUM_CHORDS + NUM_THREADS - 1)/NUM_THREADS;
        if (end > NUM_CHORDS){
            end = NUM_CHORDS;
        }
        for (int i = start; i < end; i++){
            int sum = 0;
            for(int j = 0; j < NUM_CHORDS; j++){
                sum = sum + deviceChord[i * NUM_CHORDS + j];
            }
            for (int j = 0; j < NUM_CHORDS; j++){
                deviceChord[i * NUM_CHORDS + j] = deviceChord[i * NUM_CHORDS + j] / sum;
            }
        }
    }
}

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {

    T Ncols; // --- Number of columns

    __host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

    __host__ __device__ T operator()(T i) { return i / Ncols; }
};

// If thiis works, then we'll fly with it.
// Otherwise, look into cuBLAS for doing a vector multiplication to 
// get the row sums.
void normalizeCuda(float* deviceHigh,float* deviceLow,float* deviceChord,float* highNotes,float* lowNotes,float* chords){
 
    int Nrows = NUM_NOTES * NUM_NOTES;
    int Ncols = NUM_NOTES;

    thrust::device_ptr<float> thrust_high(deviceHigh);
    // --- Allocate space for row sums and indices
    thrust::device_vector<float> d_row_sums(Nrows);
    thrust::device_vector<int> d_row_indices(Nrows);

    // --- Compute row sums by summing values with equal row indices
    thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)),
                        thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                        thrust_high,
                        d_row_indices.begin(),
                        d_row_sums.begin(),
                        thrust::equal_to<int>(),
                        thrust::plus<float>());

    thrust::reduce_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                thrust_high,
                thrust::make_discard_iterator(),
                d_row_sums.begin());

    thrust::device_ptr<float> thrust_low(deviceLow);

    // --- Compute row sums by summing values with equal row indices
    thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)),
                        thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                        thrust_low,
                        d_row_indices.begin(),
                        d_row_sums.begin(),
                        thrust::equal_to<int>(),
                        thrust::plus<float>());

    thrust::reduce_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                thrust_low,
                thrust::make_discard_iterator(),
                d_row_sums.begin());

    Nrows = NUM_CHORDS;
    Ncols = NUM_CHORDS;

    thrust::device_ptr<float> thrust_chord(deviceChord);
    thrust::device_vector<float> d_row_sums_c(Nrows);
    thrust::device_vector<int> d_row_indices_c(Nrows);

    // --- Compute row sums by summing values with equal row indices
    thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)),
                        thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                        thrust_chord,
                        d_row_indices_c.begin(),
                        d_row_sums_c.begin(),
                        thrust::equal_to<int>(),
                        thrust::plus<float>());

    thrust::reduce_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                thrust_chord,
                thrust::make_discard_iterator(),
                d_row_sums_c.begin());

    hipMemcpy(highNotes, deviceHigh, sizeof(float) * NUM_NOTES * NUM_NOTES * NUM_NOTES, hipMemcpyDeviceToHost);
    hipMemcpy(lowNotes, deviceLow, sizeof(float) * NUM_NOTES * NUM_NOTES * NUM_NOTES, hipMemcpyDeviceToHost);
    hipMemcpy(chords, deviceChord, sizeof(float) * NUM_CHORDS * NUM_CHORDS, hipMemcpyDeviceToHost);

    __syncthreads();

    hipFree(deviceHigh);
    hipFree(deviceLow);
    hipFree(deviceChord);
}
