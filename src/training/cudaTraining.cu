#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "training.h"

// Define global variables
int* deviceMajorHighNotes[NUM_GPU_PER_MATRIX];
int* deviceMajorLowNotes[NUM_GPU_PER_MATRIX];
int* deviceMajorChords;
int* deviceMinorHighNotes[NUM_GPU_PER_MATRIX];
int* deviceMinorLowNotes[NUM_GPU_PER_MATRIX];
int* deviceMinorChords;

hipStream_t majorHighStream[NUM_GPU_PER_MATRIX];
hipStream_t majorLowStream[NUM_GPU_PER_MATRIX];
hipStream_t majorChordSStream;
hipStream_t majorChordBStream;
hipStream_t minorHighStream[NUM_GPU_PER_MATRIX];
hipStream_t minorLowStream[NUM_GPU_PER_MATRIX];
hipStream_t minorChordSStream;
hipStream_t minorChordBStream;

sound_t* deviceMajorSoprano[NUM_GPU_PER_MATRIX + 1];
sound_t* deviceMajorBass[NUM_GPU_PER_MATRIX + 1];
sound_t* deviceMinorSoprano[NUM_GPU_PER_MATRIX + 1];
sound_t* deviceMinorBass[NUM_GPU_PER_MATRIX + 1];

/**
 * @brief Transforms 2 (noteTone, noteLength) pairs to a matrix index for a melody matrix,
 * and returns -1 if the row is not in the correct device
 * 
 * @param curTone tone of the current note
 * @param curDur duration of the current note
 * @param prevTone1 tone of the previous note
 * @param prevDur1 duration of the previous note
 * @param prevTone2 tone of the note before previous note
 * @param prevDur2 duration of the note before previous note
 * @param deviceI index of the GPU device we are in
 */
__device__ 
inline int findNoteCell(int curTone, int curDur, int prevTone1, int prevDur1, int prevTone2, int prevDur2, int deviceI)
{
  // current note 
  int col = curTone * NUM_DUR + curDur;

  //If previous tones are chords, get top note and find closest
  if (prevTone1 >= CHORD_OFFSET){
      prevTone1 = (prevTone1 - CHORD_OFFSET) / 144; //get top chord note
      prevTone1 = curTone - (curTone % 12) + prevTone1; //find closest prevTone1 note
  }
  if (prevTone2 >= CHORD_OFFSET){
      prevTone2 = (prevTone2 - CHORD_OFFSET) / 144; //get top chord note
      prevTone2 = curTone - (curTone % 12) + prevTone2; //find closest note
  }

  int row = ((prevTone1 * NUM_DUR) + prevDur1) * NUM_NOTES
    + ((prevTone2 * NUM_DUR) + prevDur2);

  if (row < deviceI * MATRIX_BLOCK_ROWS || row >= (deviceI + 1) * MATRIX_BLOCK_ROWS)
    return -1;

  return (row - deviceI * MATRIX_BLOCK_ROWS) * NUM_NOTES + col;
}

/**
 * @brief Transforms 2 tones into a matrix index for the chord matrix
 * 
 * @param curTone tone of the current note
 * @param prevTone tone of the previous note
 */
__device__
inline int findChordCell(int curTone, int prevTone){
	
  if (prevTone >= CHORD_OFFSET) { //is a chord
    prevTone = prevTone - CHORD_OFFSET; //shift chord down
  }
  else { //is a note
    prevTone = prevTone % 12;
  }
  
  return prevTone * NUM_CHORDS + (curTone - CHORD_OFFSET);
}

/**
 * @brief Depending on the threadIndex and GPU device index, 
 * counts note transitions in a 
 * section of the given notes array and adds
 * to existing counts on the given matrices
 * 
 * @param part notes array to count
 * @param length notes array length
 * @param deviceMatrix matrix for inserting into
 * @param deviceI index of the current GPU device
 */
__global__ 
void CountNoteSection(sound_t* part, int length, int* deviceMatrix, int deviceI)
{
  //Bounds for section to read based on threadId
  int start;
  int end;

  //Determine proper start and end bounds
  start = threadIdx.x * (length / NUM_THREADS) + 2;
  if (threadIdx.x == NUM_THREADS - 1){ //If we are the last thread, make sure to grab any extra bits
    end = length;
  }
  else{
    end = start + length/NUM_THREADS;
  } 

  //Grab previous notes and durations
  int prevTone1 = part[start-2].tone;
  int prevDur1 = part[start-2].duration;
  int curTone = part[start-1].tone;
  int curDur = part[start-1].duration;
  //Counts notes for the assigned section
  for (int noteIndex = start; noteIndex < end; noteIndex++){
    //Grab correct tones and durations for current, previous, and note before previous
  	int prevTone2 = prevTone1;
  	int prevDur2 = prevDur1;
  	prevTone1 = curTone;
    prevDur1 = curDur;
    curTone = part[noteIndex].tone;
    curDur = part[noteIndex].duration;

    if (curTone < NUM_TONES) { //if not a chord, check device and insert
      int cell = findNoteCell(curTone, curDur, prevTone1, prevDur1, prevTone2, prevDur2, deviceI);
      if (cell != -1)
        atomicAdd(&deviceMatrix[cell], 1);   
    }
  } 
}

/**
 * @brief Depending on the threadIndex, counts chord transitions in a 
 * section of the given notes array and adds
 * to existing counts on the given matrices
 * 
 * @param part notes array to count
 * @param length notes array length
 * @param deviceMatrix matrix for chords
 */
__global__ 
void CountChordSection(sound_t* part, int length, int* deviceMatrix)
{
  //Bounds for section to read based on threadId
  int start;
  int end; 

  //Determine proper start and end bounds
  start = threadIdx.x * (length / NUM_THREADS) + 2;
  if (threadIdx.x == NUM_THREADS - 1){ //If we are the last thread, make sure to grab any extra bits
    end = length;
  }
  else {
    end = start + length/NUM_THREADS;
  }

  //Grab previous notes and durations
  int prevTone1 = part[start-2].tone;
  int curTone = part[start-1].tone;
  //Counts notes for the assigned section
  for (int noteIndex = start; noteIndex < end; noteIndex++){
    //Grab correct tones and durations for current, previous, and note before previous 	
  	prevTone1 = curTone;
    curTone = part[noteIndex].tone;

    int cell; //matrix index to insert into
    if (curTone >= CHORD_OFFSET) { //insert into chord matrix
      cell = findChordCell(curTone, prevTone1);
      atomicAdd(&deviceMatrix[cell], 1);
    }
  } 
}

/**
 * @brief Allocates device memory for the matrices and initializes them,
 * as well as sets up all streams
 */
void initCuda()
{
  //Allocate device memory for all melodic line matrices
  for (int offsetGPU = 0; offsetGPU < NUM_GPU_PER_MATRIX; offsetGPU ++){

    hipSetDevice(offsetGPU + MAJOR_HIGH_DEVICE);
    hipStreamCreate(&majorHighStream[offsetGPU]);
    hipMalloc((void**)&deviceMajorSoprano[offsetGPU], sizeof(sound_t) * ARRAY_LENGTH);
    hipMalloc((void**)&deviceMajorHighNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES);
    hipMemsetAsync(deviceMajorHighNotes[offsetGPU], 0, sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, majorHighStream[offsetGPU]);

    hipSetDevice(offsetGPU + MAJOR_LOW_DEVICE);
    hipStreamCreate(&majorLowStream[offsetGPU]);
    hipMalloc((void**)&deviceMajorBass[offsetGPU], sizeof(sound_t) * ARRAY_LENGTH);
    hipMalloc((void**)&deviceMajorLowNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES);
    hipMemsetAsync(deviceMajorLowNotes[offsetGPU], 0, sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, majorLowStream[offsetGPU]);

    hipSetDevice(offsetGPU + MINOR_HIGH_DEVICE);
    hipStreamCreate(&minorHighStream[offsetGPU]);
    hipMalloc((void**)&deviceMinorSoprano[offsetGPU], sizeof(sound_t) * ARRAY_LENGTH);
    hipMalloc((void**)&deviceMinorHighNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES);
    hipMemsetAsync(deviceMinorHighNotes[offsetGPU], 0, sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, minorHighStream[offsetGPU]);

    hipSetDevice(offsetGPU + MINOR_LOW_DEVICE);
    hipStreamCreate(&minorLowStream[offsetGPU]);
    hipMalloc((void**)&deviceMinorBass[offsetGPU], sizeof(sound_t) * ARRAY_LENGTH);
    hipMalloc((void**)&deviceMinorLowNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES);
    hipMemsetAsync(deviceMinorLowNotes[offsetGPU], 0, sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, minorLowStream[offsetGPU]);
  }

  //Allocate chord matrices

  hipSetDevice(MAJOR_CHORD_DEVICE);
  hipStreamCreate(&majorChordSStream);
  hipStreamCreate(&majorChordBStream);
  hipMalloc((void**)&deviceMajorSoprano[NUM_GPU_PER_MATRIX], sizeof(sound_t) * ARRAY_LENGTH);
  hipMalloc((void**)&deviceMajorBass[NUM_GPU_PER_MATRIX], sizeof(sound_t) * ARRAY_LENGTH);
  hipMalloc((void**)&deviceMajorChords, sizeof(int) * NUM_CHORDS * NUM_CHORDS); 
  hipMemsetAsync(deviceMajorChords, 0, sizeof(int) * NUM_CHORDS * NUM_CHORDS, majorChordSStream);

  hipSetDevice(MINOR_CHORD_DEVICE);
  hipStreamCreate(&minorChordSStream);
  hipStreamCreate(&minorChordBStream);
  hipMalloc((void**)&deviceMinorSoprano[NUM_GPU_PER_MATRIX], sizeof(sound_t) * ARRAY_LENGTH);
  hipMalloc((void**)&deviceMinorBass[NUM_GPU_PER_MATRIX], sizeof(sound_t) * ARRAY_LENGTH);
  hipMalloc((void**)&deviceMinorChords, sizeof(int) * NUM_CHORDS * NUM_CHORDS); 
  hipMemsetAsync(deviceMinorChords, 0, sizeof(int) * NUM_CHORDS * NUM_CHORDS, minorChordSStream);
}

/**
 * @brief Frees device memory and streams for all matrices
 * 
 */
void freeCuda()
{
  //Step through all GPUS working on the same matrix and free
  for (int offsetGPU = 0; offsetGPU < NUM_GPU_PER_MATRIX; offsetGPU ++){

    hipSetDevice(offsetGPU + MAJOR_HIGH_DEVICE);
    hipDeviceSynchronize();
    hipStreamDestroy(majorHighStream[offsetGPU]);
    hipFree(deviceMajorSoprano[offsetGPU]);
    hipFree(deviceMajorHighNotes[offsetGPU]);

    hipSetDevice(offsetGPU + MAJOR_LOW_DEVICE);
    hipDeviceSynchronize();
    hipStreamDestroy(majorLowStream[offsetGPU]);
    hipFree(deviceMajorBass[offsetGPU]);
    hipFree(deviceMajorLowNotes[offsetGPU]);

    hipSetDevice(offsetGPU + MINOR_HIGH_DEVICE);
    hipDeviceSynchronize();
    hipStreamDestroy(minorHighStream[offsetGPU]);
    hipFree(deviceMinorSoprano[offsetGPU]);
    hipFree(deviceMinorHighNotes[offsetGPU]);

    hipSetDevice(offsetGPU + MINOR_LOW_DEVICE);
    hipDeviceSynchronize();
    hipStreamDestroy(minorLowStream[offsetGPU]);
    hipFree(deviceMinorBass[offsetGPU]);
    hipFree(deviceMinorLowNotes[offsetGPU]);
  }

  //Free on the Chord GPUs

  hipSetDevice(MAJOR_CHORD_DEVICE);
  hipDeviceSynchronize();
  hipStreamDestroy(majorChordSStream);
  hipStreamDestroy(majorChordBStream);
  hipFree(deviceMajorSoprano[NUM_GPU_PER_MATRIX]);
  hipFree(deviceMajorBass[NUM_GPU_PER_MATRIX]);
  hipFree(deviceMajorChords);
  
  hipSetDevice(MINOR_CHORD_DEVICE);
  hipDeviceSynchronize();
  hipStreamDestroy(minorChordSStream);
  hipStreamDestroy(minorChordBStream);
  hipFree(deviceMinorSoprano[NUM_GPU_PER_MATRIX]);
  hipFree(deviceMinorBass[NUM_GPU_PER_MATRIX]);
  hipFree(deviceMinorChords);
}

/**
 * @brief Stalls until all stream of the given mood are clear
 * 
 * @param mood either major or minor, denoting which streams to check
 */
void cudaStreamSynch(int mood){
  //Check all copies of melodic matrices
  for (int offsetGPU = 0; offsetGPU < NUM_GPU_PER_MATRIX; offsetGPU ++){
    if (mood == 0) { //major
      hipSetDevice(offsetGPU + MAJOR_HIGH_DEVICE);
      hipStreamSynchronize(majorHighStream[offsetGPU]);
      hipSetDevice(offsetGPU + MAJOR_LOW_DEVICE);
      hipStreamSynchronize(majorLowStream[offsetGPU]);
    }
    else { //minor
      hipSetDevice(offsetGPU + MINOR_HIGH_DEVICE);
      hipStreamSynchronize(minorHighStream[offsetGPU]);
      hipSetDevice(offsetGPU + MINOR_LOW_DEVICE);
      hipStreamSynchronize(minorLowStream[offsetGPU]);
    }
  }

  //Check chord matrix
  if (mood == 0) { //major
    hipSetDevice(MAJOR_CHORD_DEVICE);
    hipStreamSynchronize(majorChordSStream);
    hipStreamSynchronize(majorChordBStream);
  }
  else { //minor
    hipSetDevice(MINOR_CHORD_DEVICE);
    hipStreamSynchronize(minorChordSStream);
    hipStreamSynchronize(minorChordBStream);
  }
}

/**
 * @brief Counts the note transitions in the given notes arrays in parallel
 * 
 * @param soprano notes array for soprano line
 * @param sLength length of soprano array
 * @param bass notes array for bass line
 * @param bLength length of bass array
 * @param mood marks if the soprano/bass arrays are in major or minor
 */
void countTransitionsCuda(sound_t* soprano, int sLength, sound_t* bass, int bLength, int mood){

  //Determine if we should use major or minor matrices, and call GPUS to count note transitions for melodic lines
  for (int offsetGPU = 0; offsetGPU < NUM_GPU_PER_MATRIX; offsetGPU ++) {
    if (mood == 0) { //major 
      hipSetDevice(offsetGPU + MAJOR_HIGH_DEVICE);
      hipMemcpyAsync(deviceMajorSoprano[offsetGPU], soprano, sizeof(sound_t) * sLength, hipMemcpyHostToDevice, majorHighStream[offsetGPU]);
      CountNoteSection<<<1, NUM_THREADS, 0, majorHighStream[offsetGPU]>>>(deviceMajorSoprano[offsetGPU], sLength, deviceMajorHighNotes[offsetGPU], offsetGPU);
   
      hipSetDevice(offsetGPU + MAJOR_LOW_DEVICE);
      hipMemcpyAsync(deviceMajorBass[offsetGPU], bass, sizeof(sound_t) * bLength, hipMemcpyHostToDevice, majorLowStream[offsetGPU]);
      CountNoteSection<<<1, NUM_THREADS, 0, majorLowStream[offsetGPU]>>>(deviceMajorBass[offsetGPU], bLength, deviceMajorLowNotes[offsetGPU], offsetGPU);
    }
    else { //minor
      hipSetDevice(offsetGPU + MINOR_HIGH_DEVICE);
      hipMemcpyAsync(deviceMinorSoprano[offsetGPU], soprano, sizeof(sound_t) * sLength, hipMemcpyHostToDevice, minorHighStream[offsetGPU]);
      CountNoteSection<<<1, NUM_THREADS, 0, minorHighStream[offsetGPU]>>>(deviceMinorSoprano[offsetGPU], sLength, deviceMinorHighNotes[offsetGPU], offsetGPU);

      hipSetDevice(offsetGPU + MINOR_LOW_DEVICE);
      hipMemcpyAsync(deviceMinorBass[offsetGPU], bass, sizeof(sound_t) * bLength, hipMemcpyHostToDevice, minorLowStream[offsetGPU]);
      CountNoteSection<<<1, NUM_THREADS, 0, minorLowStream[offsetGPU]>>>(deviceMinorBass[offsetGPU], bLength, deviceMinorLowNotes[offsetGPU], offsetGPU);
    }
  }

  //Call GPUS for chords
  if (mood == 0) { //major
    hipSetDevice(MAJOR_CHORD_DEVICE);
    hipMemcpyAsync(deviceMajorSoprano[NUM_GPU_PER_MATRIX], soprano, sizeof(sound_t) * sLength, hipMemcpyHostToDevice, majorChordSStream);
    hipMemcpyAsync(deviceMajorBass[NUM_GPU_PER_MATRIX], bass, sizeof(sound_t) * bLength, hipMemcpyHostToDevice, majorChordBStream);
    CountChordSection<<<1, NUM_THREADS, 0, majorChordSStream>>>(deviceMajorSoprano[NUM_GPU_PER_MATRIX], sLength, deviceMajorChords);
    CountChordSection<<<1, NUM_THREADS, 0, majorChordBStream>>>(deviceMajorBass[NUM_GPU_PER_MATRIX], bLength, deviceMajorChords);
  }
  else { //minor
    hipSetDevice(MINOR_CHORD_DEVICE);
    hipMemcpyAsync(deviceMinorSoprano[NUM_GPU_PER_MATRIX], soprano, sizeof(sound_t) * sLength, hipMemcpyHostToDevice, minorChordSStream);
    hipMemcpyAsync(deviceMinorBass[NUM_GPU_PER_MATRIX], bass, sizeof(sound_t) * bLength, hipMemcpyHostToDevice, minorChordBStream);
    CountChordSection<<<1, NUM_THREADS, 0, minorChordSStream>>>(deviceMinorSoprano[NUM_GPU_PER_MATRIX], sLength, deviceMinorChords);
    CountChordSection<<<1, NUM_THREADS, 0, minorChordBStream>>>(deviceMinorBass[NUM_GPU_PER_MATRIX], bLength, deviceMinorChords);
  }
}

/**
 * @brief Copies matrices in device memory to host
 * 
 */
void cudaToHost()
{
  //Copy all melodic matrices into host memory
  for (int offsetGPU = 0; offsetGPU < NUM_GPU_PER_MATRIX; offsetGPU++) { 
    int matrixIndex = NUM_NOTES * MATRIX_BLOCK_ROWS * offsetGPU; 

    hipSetDevice(offsetGPU + MAJOR_HIGH_DEVICE);
    hipMemcpyAsync(majorHighNotes + matrixIndex, deviceMajorHighNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, hipMemcpyDeviceToHost, majorHighStream[offsetGPU]);

    hipSetDevice(offsetGPU + MAJOR_LOW_DEVICE);
    hipMemcpyAsync(majorLowNotes + matrixIndex, deviceMajorLowNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, hipMemcpyDeviceToHost, majorLowStream[offsetGPU]);

    hipSetDevice(offsetGPU + MINOR_HIGH_DEVICE);
    hipMemcpyAsync(minorHighNotes + matrixIndex, deviceMinorHighNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, hipMemcpyDeviceToHost, minorHighStream[offsetGPU]);

    hipSetDevice(offsetGPU + MINOR_LOW_DEVICE);
    hipMemcpyAsync(minorLowNotes + matrixIndex, deviceMinorLowNotes[offsetGPU], sizeof(int) * MATRIX_BLOCK_ROWS * NUM_NOTES, hipMemcpyDeviceToHost, minorLowStream[offsetGPU]);

  }

  //Copy Chord matrices
  hipSetDevice(MAJOR_CHORD_DEVICE);
  hipMemcpy(majorChords, deviceMajorChords, sizeof(int) * NUM_CHORDS * NUM_CHORDS, hipMemcpyDeviceToHost);

  hipSetDevice(MINOR_CHORD_DEVICE);
  hipMemcpy(minorChords, deviceMinorChords, sizeof(int) * NUM_CHORDS * NUM_CHORDS, hipMemcpyDeviceToHost);
}
